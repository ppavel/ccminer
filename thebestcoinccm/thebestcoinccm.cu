// Copyright (c) 2017 TheBestCoin developers
//
// Based on original work by sp and DJM34.
//
// This program is free software: you can redistribute it and/or modify
// it under the terms of the GNU General Public License as published by
// the Free Software Foundation, either version 2 of the License, or
// (at your option) any later version. See COPYING for more details.

extern "C" {
#include "sph/sph_blake.h"
#include "sph/sph_bmw.h"
#include "sph/sph_skein.h"
#include "sph/sph_keccak.h"
#include "sph/sph_cubehash.h"
}

#include "miner.h"
#include "cuda_helper.h"
#include "cuda_utils.h"
#include "cuda_lyra2v2_var.h"

#define DEFAULT_INTENSITY 16384
#define MAX_INTENSITY_WIN 65536

static _ALIGN(64) uint64_t *d_hash[MAX_GPUS] = { NULL };
static  uint64_t *d_hash2[MAX_GPUS] = { NULL };

extern void blakeKeccak256_cpu_hash_80(const int thr_id, const uint32_t threads, const uint32_t startNonce, uint64_t *Hash);
extern void blake256_cpu_hash_80(int thr_id, const uint32_t threads, const uint32_t startNonce, uint64_t *Hash);
extern void Keccak256_cpu_hash_32(int thr_id, const uint32_t threads, const uint32_t startNonce, uint64_t *Hash);
extern void blake256_cpu_setBlock_80(uint32_t *pdata);

extern void keccak256_cpu_hash_32(int thr_id, uint32_t threads, uint32_t startNonce, uint64_t *d_outputHash);
extern void keccak256_cpu_init(int thr_id, uint32_t threads);

extern void skein256_cpu_hash_32(int thr_id, uint32_t threads, uint32_t startNonce, uint64_t *d_outputHash);
extern void skein256_cpu_init(int thr_id, uint32_t threads);

extern void skeinCube256_cpu_hash_32(int thr_id, uint32_t threads, uint32_t startNonce, uint64_t *d_outputHash);

extern void bmw256_cpu_init(int thr_id, uint32_t threads);
extern void bmw256_cpu_hash_32(int thr_id, uint32_t threads, uint32_t startNounce, uint64_t *g_hash, uint32_t *resultnonces, uint32_t target);

extern void cubehash256_cpu_hash_32(int thr_id, uint32_t threads, uint32_t startNounce, uint64_t *d_hash);

extern "C" int LYRA2(void *K, uint64_t kLen, const void *pwd, uint64_t pwdlen, const void *salt, uint64_t saltlen, uint64_t timeCost, uint64_t nRows, uint64_t nCols);

/**
 * CPU version of TheBestCoin algorithm
 */
extern "C" void thebestcoinccm_hash(void *state, const void *input)
{
	sph_blake256_context      ctx_blake;
	sph_keccak256_context     ctx_keccak;
	sph_skein256_context      ctx_skein;
	sph_bmw256_context        ctx_bmw;
	sph_cubehash256_context   ctx_cube;

	uint32_t hashA[8], hashB[8];

	sph_blake256_init(&ctx_blake);
	sph_blake256(&ctx_blake, input, 80);
	sph_blake256_close(&ctx_blake, hashA);

	sph_keccak256_init(&ctx_keccak);
	sph_keccak256(&ctx_keccak, hashA, 32);
	sph_keccak256_close(&ctx_keccak, hashB);

	sph_cubehash256_init(&ctx_cube);
	sph_cubehash256(&ctx_cube, hashB, 32);
	sph_cubehash256_close(&ctx_cube, hashA);


	LYRA2(hashB, 32, hashA, 32, hashA, 32, 1, 8, 32);

	sph_skein256_init(&ctx_skein);
	sph_skein256(&ctx_skein, hashB, 32);
	sph_skein256_close(&ctx_skein, hashA);

	sph_cubehash256_init(&ctx_cube);
	sph_cubehash256(&ctx_cube, hashA, 32);
	sph_cubehash256_close(&ctx_cube, hashB);


	sph_bmw256_init(&ctx_bmw);
	sph_bmw256(&ctx_bmw, hashB, 32);
	sph_bmw256_close(&ctx_bmw, hashA);

	memcpy(state, hashA, 32);
}

static bool init[MAX_GPUS] = { 0 };

extern "C" int scanhash_thebestcoinccm(int thr_id, uint32_t *pdata,
	const uint32_t *ptarget, uint32_t max_nonce,
	unsigned long *hashes_done)
{
	const uint32_t first_nonce = pdata[19];
	const size_t msize = BLOCK_LEN_BYTES * 32 * 8; // Matrix size

	uint32_t intensity = 16384;
	uint32_t tpb = 8;

	hipDeviceProp_t props = { 0 };
	hipGetDeviceProperties(&props, device_map[thr_id]);

	// Values of tpb and intensity can be changed for specific video card to tune performance
	//if (strstr(props.name, "980 Ti"))
	//{
	//	tpb = 10;
	//	intensity = 256 * 16 * 18;
	//}
	//else
	//{
		const size_t mvram = size_t(props.totalGlobalMem / 5) * 4;
		intensity = roundown2(mvram / msize);
		if (!intensity)
		{
			intensity = DEFAULT_INTENSITY;
		}
#ifdef WIN32
		else if (intensity > MAX_INTENSITY_WIN)
		{
			intensity = MAX_INTENSITY_WIN;
		}
#endif
	//}

	uint32_t throughput = device_intensity(device_map[thr_id], __func__, intensity);

	if (opt_benchmark)
		((uint32_t*)ptarget)[7] = 0x00ff;
	if (!init[thr_id])
	{
		hipSetDevice(device_map[thr_id]);
		if (!opt_cpumining) hipSetDeviceFlags(hipDeviceScheduleBlockingSync);
		if (opt_n_gputhreads == 1)
		{
			hipDeviceSetCacheConfig(hipFuncCachePreferL1);
		}
		skein256_cpu_init(thr_id, throughput);
		bmw256_cpu_init(thr_id, throughput);

		if (!d_hash2[thr_id])
			CUDA_SAFE_CALL(hipMalloc(&d_hash2[thr_id], ROW_LEN_BYTES * 8 * size_t(throughput)));
		if (!d_hash[thr_id])
			CUDA_SAFE_CALL(hipMalloc(&d_hash[thr_id], 8 * sizeof(uint32_t) * size_t(throughput)));

		lyra2v2_cpu_init_VAR(thr_id, throughput, d_hash2[thr_id]);

		init[thr_id] = true;
	}

	uint32_t endiandata[20];
	for (int k = 0; k < 20; k++)
		be32enc(&endiandata[k], ((uint32_t*)pdata)[k]);

	blake256_cpu_setBlock_80(pdata);

	do
	{
		uint32_t foundNonce[2] = { 0, 0 };

		blakeKeccak256_cpu_hash_80(thr_id, throughput, pdata[19], d_hash[thr_id]);
		ON_CUDA_ERROR_BREAK;

		cubehash256_cpu_hash_32(thr_id, throughput, pdata[19], d_hash[thr_id]);
		ON_CUDA_ERROR_BREAK;

		lyra2v2_cpu_hash_32_VAR(thr_id, throughput, pdata[19], d_hash[thr_id], tpb);
		ON_CUDA_ERROR_BREAK;

		skein256_cpu_hash_32(thr_id, throughput, pdata[19], d_hash[thr_id]);
		ON_CUDA_ERROR_BREAK;

		cubehash256_cpu_hash_32(thr_id, throughput, pdata[19], d_hash[thr_id]);
		ON_CUDA_ERROR_BREAK;

		bmw256_cpu_hash_32(thr_id, throughput, pdata[19], d_hash[thr_id], foundNonce, ptarget[7]);
		ON_CUDA_ERROR_BREAK;

		if (foundNonce[0] != 0xffffffff)
		{
			const uint32_t Htarg = ptarget[7];
			uint32_t vhash64[8];
			be32enc(&endiandata[19], foundNonce[0]);
			thebestcoinccm_hash(vhash64, endiandata);
			if (vhash64[7] <= Htarg && fulltest(vhash64, ptarget))
			{
				int res = 1;
				// check if there was some other ones...
				*hashes_done = pdata[19] - first_nonce + throughput;
				if (foundNonce[1] != 0xffffffff)
				{
					pdata[21] = foundNonce[1];
					res++;
					if (opt_benchmark)  applog(LOG_INFO, "GPU #%d Found second nounce %08x", thr_id, foundNonce[1], vhash64[7], Htarg);
				}
				pdata[19] = foundNonce[0];
				if (opt_benchmark) applog(LOG_INFO, "GPU #%d Found nounce % 08x", thr_id, foundNonce[0], vhash64[7], Htarg);
				return res;
			}
			else
			{
				if (vhash64[7] > Htarg) // don't show message if it is equal but fails fulltest
					applog(LOG_WARNING, "GPU #%d: result does not validate on CPU!", thr_id);
			}
		}

		pdata[19] += throughput;

	}
	while (!scan_abort_flag && !work_restart[thr_id].restart && ((uint64_t)max_nonce > ((uint64_t)(pdata[19]) + (uint64_t)throughput)));

	*hashes_done = pdata[19] - first_nonce;
	return 0;
}
